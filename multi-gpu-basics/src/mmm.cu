#ifndef MMM_H
#define MMM_H
#include <iostream> // for debuggin
    
namespace singleGPU {

    template <class ElTp, int T> 
    __global__ void matMultRegTiledKernel(
        ElTp* A,
        ElTp* B,
        ElTp* C, 
        int heightA, 
        int widthB, 
        int widthA
    ) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  bidy * T;
        //int const bdimx = blockDim.x; // =Tile
        //int const bdimy = blockDim.y; // =Tile

        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + bidy * T < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
            __syncthreads();


            for(int i = 0; i < T; i++){
                if ((ii + i) < heightA && j < widthB)  {
                    C[(i + ii)*widthB + j] = Creg[i];
                }
            }
        }
    }



    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {
            dim3 block(T, T, 1);
            int grid_x = ceil((float)B_width / (T * T));
            int grid_y = ceil((float)A_height / (T)); 
            dim3 grid(grid_x, grid_y, 1);


            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);
            return hipGetLastError();
    }

    template<class ElTp, int T>
    __global__ void matMultTrivial(ElTp* A, ElTp* B, ElTp* C, int A_height, int B_width, int B_height){
        const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t j = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (i < B_width || j < A_height) {
            int accum = 0;
            for(int k = 0; k < B_height; k++){
                accum += A[j*B_height + k] * B[k*B_width + i];
            }
            C[j * A_height + i] = accum;
        }
    }

    template< class ElTp, int T>
    hipError_t MMM_trivial(
        ElTp* A,
        ElTp* B, 
        ElTp* C, 
        int A_height, 
        int B_width, 
        int B_height
    ) {
        dim3 block(T, T, 1);
        int grid_x = ceil((float)B_width / (T));
        int grid_y = ceil((float)A_height / (T)); 
        dim3 grid(grid_x, grid_y, 1);

        matMultTrivial< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);


        return hipPeekAtLastError();
    }


}    

namespace multiGPU {

    template <class ElTp, int T> 
    __global__ void matMultRegTiledKernel(ElTp* A, ElTp* B, ElTp* C, int heightA, int widthB, int widthA, int devID) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  gridDim.y * T * devID + bidy * T;


        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + ii < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
        }
        __syncthreads();
        for(int i = 0; i < T; i++){
            if ((ii + i) < heightA && j < widthB)  {
                C[(i + ii) * widthB + j] = Creg[i];
            }
        }
    }


    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {

        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);


        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + DeviceCount - 1) / DeviceCount; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);

        for(int dev_id = 0; dev_id < DeviceCount; dev_id++){
            hipSetDevice(dev_id);
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);

        }
        return hipGetLastError();
    }

    template< class ElTp, int T>
    hipError_t MMM_emulated(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height,
            int emulatedDevices
        ) {
        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + emulatedDevices - 1) / emulatedDevices; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);


        for(int dev_id = 0; dev_id < emulatedDevices; dev_id++){
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        return hipGetLastError();
    }       

    template<class ElTp, int T>
    __global__ void matMultTrivial(ElTp* A, ElTp* B, ElTp* C, int A_height, int B_width, int B_height, int devID){
        const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t j = devID * gridDim.y * blockDim.y  + blockIdx.y * blockDim.y + threadIdx.y;
        
        if (i < B_width || j < A_height) {
            int accum = 0;
            for(int k = 0; k < B_height; k++){
                accum += A[j*B_height + k] * B[k*B_width + i];
            }
            C[j * A_height + i] = accum;
        }
    }

    template< class ElTp, int T>
    hipError_t MMM_trivial_emulated(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height,
            int emulatedDevices
        ) {
        dim3 block(T, T, 1);
        //std::cout << A_height << ", " << B_width << ", " << B_height << ", " << T <<  "\n";

        int grid_x_total = ceil((float)B_width / (T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + emulatedDevices - 1) / emulatedDevices; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);


        for(int dev_id = 0; dev_id < emulatedDevices; dev_id++){
            matMultTrivial< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        return hipGetLastError();
    } 

}

#endif